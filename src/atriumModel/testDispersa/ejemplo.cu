
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>

int assert2(bool t, char *msg){
    if(t){
        printf("todo bien %s\n", msg);
    }
    else{
        printf("fail %s\n",msg);
    }
    return 0;
}


int main(){
    hipSetDevice(0);
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipsolverSpHandle_t cusolverH;
    hipError_t cudaStatus;
    hipsparseStatus_t cusparse_status;// = CUSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status;//= CUSOLVER_STATUS_SUCCESS;

    hipsparseMatDescr_t descrA;

    int *d_csrRowPtrA;
    int reorder = 0;
    int singularity = 0;
    double tol = 0.00001;
    int *d_csrColIndA;
    double *d_csrValA;
    double *d_b;
    double *d_x;
    double *x;


/*    | 1     0    0   0 |
      | 0     2    0   0 |
  A=  | 0     0    3   0 |
      | 0.1  0.1  0.1  4 |
  CSR of A is based-1

  b = [1 1 1 1 ]
*/

    const int m = 4; // row size
    const int n = 4; // nxn matrix A
    const int nnzA = 7; // number of non-zero elements in A
    /*const int csrRowPtrA[m+1] = {1, 2, 3, 4, 8};
    const int csrColIndA[nnzA] = {1, 2, 3, 1, 2, 3, 4};
    const double csrValA[nnzA] = {1.0, 2.0, 3.0, 0.1, 0.1, 0.1, 4.0};
    const double b[m] = {1.0, 1.0, 1.0, 1.0};*/

    int csrRowPtrA[m+1];
    csrRowPtrA[0] = 1;
    csrRowPtrA[1] = 2;
    csrRowPtrA[2] = 3;
    csrRowPtrA[3] = 4;
    csrRowPtrA[4] = 8;

    int csrColIndA[nnzA];
    csrColIndA[0] = 1;
    csrColIndA[1] = 2;
    csrColIndA[2] = 3;
    csrColIndA[3] = 1;
    csrColIndA[4] = 2;
    csrColIndA[5] = 3;
    csrColIndA[6] = 4;

    double csrValA[nnzA];
    csrValA[0] = 1.0;
    csrValA[1] = 2.0;
    csrValA[2] = 3.0;
    csrValA[3] = 0.1;
    csrValA[4] = 0.1;
    csrValA[5] = 0.1;
    csrValA[6] = 4.0;

    double b[m];
    b[0] = 1.0;
    b[1] = 1.0;
    b[2] = 1.0;
    b[3] = 1.0;

    x = (double *)malloc(m*sizeof(double));

    // Create cusolver handle, qr info and matrix descriptor
    cusolver_status = hipsolverSpCreate(&cusolverH);
    assert2(cusolver_status == HIPSOLVER_STATUS_SUCCESS, "hipsolverSpCreate");


    cusolver_status = hipsolverSpSetStream(cusolverH, stream1);
    assert2(cusolver_status == HIPSOLVER_STATUS_SUCCESS, "Assign cusolver stream");

    cusparse_status = hipsparseCreateMatDescr(&descrA);
    assert2(cusparse_status == HIPSPARSE_STATUS_SUCCESS, "hipsparseCreateMatDescr");

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    hipDeviceSynchronize();



    // copy A and b to device
    cudaStatus = hipMalloc((void**)&d_csrValA, sizeof(double)*nnzA);
    assert2(cudaStatus == hipSuccess, "hipMalloc d_csrValA");

    cudaStatus = hipMalloc((void**)&d_csrColIndA, sizeof(int)*nnzA);
    assert2(cudaStatus == hipSuccess, "hipMalloc d_csrColIndA");

    cudaStatus = hipMalloc((void**)&d_csrRowPtrA, sizeof(int)*(m+1));
    assert2(cudaStatus == hipSuccess, "hipMalloc d_csrRowPtrA");

    cudaStatus = hipMalloc((void**)&d_b, sizeof(double)*m);
    assert2(cudaStatus == hipSuccess,"hipMalloc d_b");

    cudaStatus = hipMalloc((void**)&d_x, sizeof(double)*m);
    assert2(cudaStatus == hipSuccess,"hipMalloc d_x");

    cudaStatus = hipMemcpy(d_csrValA, csrValA, sizeof(double)*nnzA, hipMemcpyHostToDevice);
    assert2(cudaStatus == hipSuccess,"hipMemcpy csrValA");

    cudaStatus = hipMemcpy(d_csrColIndA, csrColIndA, sizeof(int)*nnzA, hipMemcpyHostToDevice);
    assert2(cudaStatus == hipSuccess,"hipMemcpy csrColIndA");

    cudaStatus = hipMemcpy(d_csrRowPtrA, csrRowPtrA, sizeof(int)*(m+1), hipMemcpyHostToDevice);
    assert2(cudaStatus == hipSuccess,"hipMemcpy csrRowPtrA");

    cudaStatus = hipMemcpy(d_b, b, sizeof(double)*m, hipMemcpyHostToDevice);
    assert2(cudaStatus == hipSuccess,"cudamemcpy b");


    cusolver_status = hipsolverSpDcsrlsvqr(cusolverH, m , nnzA, descrA, d_csrValA,
            d_csrRowPtrA, d_csrColIndA, d_b, tol, reorder, d_x, &singularity);

    assert2(cusolver_status == HIPSOLVER_STATUS_SUCCESS,"Call hipsolverSpDcsrlsvqr");
    hipDeviceSynchronize();

    cudaStatus = hipMemcpy(x, d_x, sizeof(double)*m, hipMemcpyDeviceToHost);
    assert2(cudaStatus == hipSuccess,"hipMemcpy d_x");

    hipFree(d_csrValA);hipFree(d_csrColIndA);hipFree(d_csrRowPtrA);hipFree(d_b);hipFree(d_x);
    free(x);

    return 0;

}
